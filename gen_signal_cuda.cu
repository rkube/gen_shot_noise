#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include "include/gen_signal.h"

using namespace std;

__global__ void add_to_signal(int* d_burst_tidx, 
                              double* d_burst_amplitude, 
                              int K, 
                              double* d_signal, 
                              int N,
                              int g_offset, 
                              double dt, 
                              double l){
// Compute the total offsets this thread has to compute:
    const int thread_offset = blockIdx.x * blockDim.x + threadIdx.x;
    const int offset = thread_offset + g_offset;

    int k = 0;
    double result = 0.0;
    double bexp = 0.0;
    for(k = 0; k < K; k++){
        if (offset < d_burst_tidx[k]){
            bexp = dt * (offset - d_burst_tidx[k]) / l;
            //result += d_burst_amplitude[k] * exp(dt * (offset - d_burst_tidx[k]) / l);
        } else if(offset >= d_burst_tidx[k]) {
            bexp = dt * (d_burst_tidx[k] - offset) / (1.0 - l);
            //result += d_burst_amplitude[k] * exp(dt * (d_burst_tidx[k] - offset) / (1.0 - l));
        }
        result += d_burst_amplitude[k] * exp(bexp);
    }
    d_signal[thread_offset] = result;
}

void generate_ts_cuda(int* burst_tidx, double* burst_amplitude, int K,
                      double* signal, int N, double dt, double l){
    int i_it = 0;
    int round_t_offset = 0;
    size_t size_K_double = 0; 
    size_t size_K_int = 0;
    size_t size_N = 0;

    // Pointers to device memory 
    int* d_burst_tidx;
    double* d_burst_amplitude;
    double* d_signal;
    hipError_t err;

    // Burst parameters that fall in round when wrapping is used
    vector<int> round_burst_tidx;
    vector<double> round_burst_amplitude;

    // Crop signal if it is too long
    const size_t signal_size = N * sizeof(double);
    if (signal_size > cuda_max_gpu_mem){
        cerr << "Array does not fit into GPU memory: " << N * sizeof(double) << "bytes requested\n";
        cerr << "Max allowed: " << cuda_max_gpu_mem << " bytes\n";
        cerr << "Truncating: " << N << " -> " << cuda_max_gpu_mem / sizeof(double) << " elements \n";
        N = cuda_max_gpu_mem / sizeof(double);
    }

    // Elements in time series process per kernel call
    const int elem_per_round = cuda_num_blocks * cuda_blocksize;

    // Compute total number of blocks to be computed
    const int total_blocks = N / cuda_blocksize;
    // Number of rounds considering cuda_max_blocks
    const int num_rounds = total_blocks / cuda_num_blocks;
    // Overlap expressed in tidx
    const int round_halo = 300;

    cout << "N = " << N << ", " << cuda_blocksize << "elements per block, ";
    cout << cuda_num_blocks << "blocks per round, " << num_rounds << " rounds\n";

    int round_num_bursts = 0;
    // Limit GPU Kernel to have cuda_max_blocks blocks per call
    for(int round = 0; round < num_rounds; round++){
        // Flush burst parameters for current round
        round_burst_tidx.clear();
        round_burst_amplitude.clear();
        // Compute offset for current round
        round_t_offset = round * elem_per_round;
        // Find burst arrival times falling in the current bounds. Include those burst
        // arrival times falling in the block overlap from the previous round
        for(i_it = 0; i_it < K; i_it++){
            if ((burst_tidx[i_it] > round_t_offset - elem_per_round * round_halo) &&
                 (burst_tidx[i_it] < round_t_offset + elem_per_round))
            {
                //cout << "Burst: t = " << burst_tidx[i_it] << ", Amplitude: " << burst_amplitude[i_it] << "\n";
                round_burst_tidx.push_back(burst_tidx[i_it]);
                round_burst_amplitude.push_back(burst_amplitude[i_it]);
            }
        }
        //cout << round_burst_tidx.size() << " past bursts in round " << round << "/" << num_rounds << "\n";
        // Bursts to treat in this round
        round_num_bursts = round_burst_tidx.size();
        size_K_int = round_num_bursts * sizeof(int);
        size_K_double = round_num_bursts * sizeof(double);
        size_N = elem_per_round * sizeof(double);
        // Allocate device memory and copy burst parameters to device
        if ( (err = hipMalloc(&d_burst_tidx, size_K_int)) != hipSuccess ){
            cerr << "hipMalloc failed for " << size_K_int << "bytes: " << hipGetErrorString(err) << "\n";
            exit(1);
        }
        if ( (err = hipMalloc(&d_burst_amplitude, size_K_double)) != hipSuccess ){
            cerr << "hipMalloc failed for " << size_K_double << "bytes: " << hipGetErrorString(err) << "\n";
            exit(1);
        }
        if ( (err = hipMalloc(&d_signal, size_N)) != hipSuccess ){
            cerr << "hipMalloc failed for " << size_N << "bytes: " << hipGetErrorString(err) << "\n";
            exit(1);
        }
        hipMemcpy(d_burst_tidx, round_burst_tidx.data(), size_K_int, hipMemcpyHostToDevice);
        hipMemcpy(d_burst_amplitude, round_burst_amplitude.data(), size_K_double, hipMemcpyHostToDevice);
        //hipMemcpy(d_signal, signal + round_t_offset, size_N, hipMemcpyHostToDevice);

        add_to_signal<<<cuda_num_blocks, cuda_blocksize>>>(d_burst_tidx, d_burst_amplitude, round_num_bursts, d_signal, elem_per_round, round_t_offset, dt, l);
        hipMemcpy(signal + round_t_offset, d_signal, size_N, hipMemcpyDeviceToHost);

        hipFree(d_signal);
        hipFree(d_burst_amplitude);
        hipFree(d_burst_tidx);
    }

}


void generate_ts_cuda_v2(vector<pulse>& pulses, double dt, double* signal)
{


}
// End of file gen_signal_cuda.cu
